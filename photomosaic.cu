#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "photomosaic.h"

struct GlobalConstants {
  int numImages;
  int finalIndex;
  int numSlices;
  int cutSize;

  float* imageAverages;
  float* allAverages;
  int* imageIndex;
};

__constant__ GlobalConstants cuConstMosaicParams;


__global__ void kernelMatchImages() {

}

CudaMosaic::CudaMosaic() {
  numImages = 0;
  finalIndex = 0;
  numSlices = 0;
  cutSize = 0;

  cudaDeviceImageAverages = NULL;
  cudaDeviceAllAverages = NULL;
  cudaDeviceImageIndex = NULL;

  imageAverages = NULL;
  allAverages = NULL;
  imageIndex = NULL;
}

CudaMosaic::~CudaMosaic() {
  if (imageAverages) {
    delete [] imageAverages;
    delete [] allAverages;
    delete [] imageIndex;
  }

  if (cudaDeviceImageAverages) {
    delete [] cudaDeviceImageAverages;
    delete [] cudaDeviceAllAverages;
    delete [] cudaDeviceImageIndex;
  }
}

void CudaMosaic::setup() {
  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("Initializing CUDA for CudaRenderer\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i=0; i<deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }

  hipMalloc(&cudaDeviceImageAverages, sizeof(float) * cutSize * 3 * (numSlices * numSlices));
  hipMalloc(&cudaDeviceAllAverages, sizeof(float) * cutSize * 3 * numImages);
  hipMalloc(&cudaDeviceImageIndex, sizeof(int) * numImages);

  hipMemcpy(cudaDeviceImageAverages, imageAverages, sizeof(float) * cutSize * 3 * (numSlices * numSlices), hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceAllAverages, allAverages, sizeof(float) * cutSize * 3 * numImages, hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceImageIndex, imageIndex, sizeof(int) * numImages, hipMemcpyHostToDevice);

  GlobalConstants params;
  params.numImages = numImages;
  params.finalIndex = finalIndex;
  params.numSlices = numSlices;
  params.cutSize = cutSize;
  params.imageAverages = cudaDeviceImageAverages;
  params.allAverages = cudaDeviceAllAverages;
  params.imageIndex = cudaDeviceImageIndex;

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstMosaicParams), &params, sizeof(GlobalConstants));
}

void CudaMosaic::imageMatch() {
  dim3 threadsPerBock(cutSize, cutSize, 1);
  dim3 numBlocks(numSlices, numSlices, 1);

  kernelMatchImages<<<numBlocks, threadsPerBock>>>();
}















